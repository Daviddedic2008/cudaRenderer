#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_math_constants.h"
#include <time.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string>

#define scr_w 512
#define scr_h 512

#define num_bounces 1
#define num_frames 100

__inline__ __device__ float3 init_float3(float x, float y, float z) {
	float3 ret;
	ret.x = x;
	ret.y = y;
	ret.z = z;
	return ret;
}

__inline__ __device__ float3 add_float3(float3 v1, float3 v2) {
	v1.x += v2.x;
	v1.y += v2.y;
	v1.z += v2.z;
	return v1;
}

__inline__ __device__ float3 sub_float3(float3 v1, float3 v2) {
	v1.x -= v2.x;
	v1.y -= v2.y;
	v1.z -= v2.z;
	return v1;
}

__inline__ __device__ float3 mult_float3(float3 v1, float3 v2) {
	v1.x *= v2.x;
	v1.y *= v2.y;
	v1.z *= v2.z;
	return v1;
}

__inline__ __device__ float3 scale_float3(float3 v1, float scl) {
	v1.x *= scl;
	v1.y *= scl;
	v1.z *= scl;
	return v1;
}

#define dot(float3_v1, float3_v2) (float3_v1.x * float3_v2.x + float3_v1.y * float3_v2.y + float3_v1.z * float3_v2.z)
#define matrix2D_eval(float_a , float_b, float_c, float_d) (float_a*float_d - float_b*float_c)
#define matgnitude(float3_a) (sqrtf(dot(float3_a, float3_a)))

__inline__ __device__ float3 cross_float3(float3 v1, float3 v2) {
	float3 ret;
	ret.x = matrix2D_eval(v1.y, v1.z, v2.y, v2.z);
	ret.y = matrix2D_eval(v1.x, v1.z, v2.x, v2.z);
	ret.z = matrix2D_eval(v1.x, v1.y, v2.x, v2.y);
	return ret;
}

__inline__ __device__ float3 normalize_float3(float3 v1) {
	float scl = 1 / sqrtf(dot(v1, v1));
	v1 = scale_float3(v1, scl);
	return v1;
}

__inline__ __device__ float3 invert_float3(float3 f) {
	f.x *= -1;
	f.y *= -1;
	f.z *= -1;
	return f;
}

__inline__ __device__ float3 norm_float3(float3 p1, float3 p2, float3 p3) {
	return cross_float3(sub_float3(p1, p2), sub_float3(p1, p3));
}

__inline__ __device__ float3 rand_float3(unsigned int seed) {
	float3 ret;
	seed ^= seed << 13;
	seed ^= seed >> 17;
	seed ^= seed << 5;
	ret.x = (seed % 10000) / 10000.0f;
	seed ^= seed * seed;
	ret.y = (seed % 10000) / 10000.0f;
	seed ^= seed * seed;
	ret.z = (seed % 10000) / 10000.0f;
	return ret;
}

__inline__ __device__ float3 rand_offset_float3(float3 v, float3 norm, float strength, unsigned int seed) {
	float3 rnd = rand_float3(seed);
	float3 axis = cross_float3(rnd, norm);
	seed ^= seed << 13;
	seed ^= seed >> 17;
	seed ^= seed << 5;
	float scl = (seed % 1000) / 1000.0f;
	axis = scale_float3(axis, scl * strength);
	v = scale_float3(v, 1.0f - strength);
	return add_float3(v, axis);
}

typedef struct {
	float r, g, b;
}color;

__device__ color device_color_buffer[scr_w * scr_h];
color color_buffer[scr_w * scr_h];

__device__ float max_brightness_buffer[scr_w * scr_h];

color ray_color_buffer[scr_w * scr_h];

__device__ __inline__ color init_color(float r, float g, float b) {
	color ret;
	ret.r = r;
	ret.g = g;
	ret.b = b;
	return ret;
}

typedef struct {
	color c;
	float brightness, roughness;
}material;

__inline__ __device__ material init_material(float r, float g, float b, float roughness, float brightness) {
	material ret;
	ret.c = init_color(r, g, b);
	ret.brightness = brightness;
	ret.roughness = roughness;
	return ret;
}

typedef struct {
	float3 origin, direction;
	float3 last_intersect;
	float last_dist;
	int last_triangle_index;
	bool hit_triangle;
	unsigned char num_intersects;
}ray;

__device__ ray rays[scr_w * scr_h];
#define fov 0.02f

__global__ void init_rays() {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int x, y;
	x = id % scr_w;
	y = id / scr_w;
	ray r;
	r.origin.x = x;
	r.origin.y = y;
	r.origin.z = 0.0f;
	r.direction.x = x * fov;
	r.direction.y = y * fov;
	r.direction.z = 1.0f;
	r.num_intersects = 0;
	device_color_buffer[id] = init_color(0.0f, 0.0f, 0.0f);
	max_brightness_buffer[id] = 0.0f;
	rays[id] = r;
}

__global__ void reset_rays() {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	rays[id].last_dist = -1.0f;
	rays[id].last_triangle_index = -1;
	rays[id].hit_triangle = false;
}

void init_rays_call() {
	init_rays << <1024, scr_w* scr_h / 1024 >> > ();
}

void reset_rays_call() {
	reset_rays << <1024, scr_w* scr_h / 1024 >> > ();
}

#pragma 
typedef struct {
	float3 p1, p2, p3;
	float3 nv;
	float3 sb21, sb31;
	float dot2121, dot2131, dot3131;
}triangle;

#define num_triangles 12
#define triangles_per_load 256

__device__ triangle triangles[num_triangles];
__device__ material triangle_materials[num_triangles];

__inline__ __device__ triangle init_triangle(float3 p1, float3 p2, float3 p3) {
	triangle ret;
	ret.p1 = p1;
	ret.p2 = p2;
	ret.p3 = p3;
	ret.nv = norm_float3(p1, p2, p3);
	ret.sb21 = sub_float3(p2, p1);
	ret.sb31 = sub_float3(p3, p1);
	ret.dot2121 = dot(ret.sb21, ret.sb21);
	ret.dot2131 = dot(ret.sb31, ret.sb21);
	ret.dot3131 = dot(ret.sb31, ret.sb31);
	return ret;
}

__global__ void init_cube(float3 p1, float l, int index, float r, float g, float b, float br, float roughness) {
	// Compute the 8 vertices of the cube
	float3 p2 = make_float3(p1.x + l, p1.y, p1.z);
	float3 p3 = make_float3(p1.x, p1.y - l, p1.z);
	float3 p4 = make_float3(p1.x + l, p1.y - l, p1.z);
	float3 p5 = make_float3(p1.x, p1.y, p1.z - l);
	float3 p6 = make_float3(p1.x + l, p1.y, p1.z - l);
	float3 p7 = make_float3(p1.x, p1.y - l, p1.z - l);
	float3 p8 = make_float3(p1.x + l, p1.y - l, p1.z - l);

	triangles[index] = init_triangle(p1, p3, p2);       // Front face
	triangles[index + 1] = init_triangle(p2, p3, p4);
	triangles[index + 2] = init_triangle(p5, p6, p7);   // Back face
	triangles[index + 3] = init_triangle(p6, p8, p7);
	triangles[index + 4] = init_triangle(p1, p5, p3);   // Left face
	triangles[index + 5] = init_triangle(p5, p7, p3);
	triangles[index + 6] = init_triangle(p2, p4, p6);   // Right face
	triangles[index + 7] = init_triangle(p6, p4, p8);
	triangles[index + 8] = init_triangle(p1, p2, p5);   // Top face
	triangles[index + 9] = init_triangle(p2, p6, p5);
	triangles[index + 10] = init_triangle(p3, p7, p4);  // Bottom face
	triangles[index + 11] = init_triangle(p7, p8, p4);
	for (int a = 0; a < 12; a++) {
		triangle_materials[index + a] = init_material(r, g, b, roughness, br);
	}
}

void init_cube_CPU(float3 p1, float l, int index, float r, float g, float b, float bright, float rough) {
	init_cube << <1, 1 >> > (p1, l, index, r, g, b, bright, rough);
}

__global__ void set_triangle_kernel(float3 p1, float3 p2, float3 p3, int index, float R, float G, float B, float b, float r) {
	triangles[index] = init_triangle(p1, p2, p3);
	triangle_materials[index] = init_material(R, G, B, r, b);
}

void set_triangle(float3 p1, float3 p2, float3 p3, int index) {
	set_triangle_kernel << <1, 1 >> > (p1, p2, p3, index, 1.0f, 0.0f, 0.0f, 1.0f, 0.0f);
}

#define threads_bounce 512
#define blocks_bounce scr_w * scr_h / threads_bounce

__constant__ triangle tempLoader[triangles_per_load];

__global__ void bounceKernel(int iteration) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int passes = num_triangles / triangles_per_load + 1;
	ray r = rays[index];
	for (int p = 0; p < passes; p++) {
		int tid = p * triangles_per_load + index;
		if (tid < num_triangles && tid < triangles_per_load) {
			tempLoader[threadIdx.x] = triangles[tid];
		}
		__syncthreads();
		float tmp = p * triangles_per_load;
		
#pragma unroll
		for (int ti = 0; ti < triangles_per_load && (ti + tmp) < num_triangles; ti++) {
			triangle t = tempLoader[ti];
			float disc = dot(r.direction, t.nv);
			if (disc == 0.0f) { continue; }
			float3 temp_sub;
			temp_sub.x = t.p1.x - r.origin.x;
			temp_sub.y = t.p1.y - r.origin.y;
			temp_sub.z = t.p1.z - r.origin.z;
			float tmp_dt = __fdividef(dot(t.nv, temp_sub), disc);
			float3 intersect;
			intersect.x = r.origin.x + r.direction.x * tmp_dt;
			intersect.y = r.origin.y + r.direction.y * tmp_dt;
			intersect.z = r.origin.z + r.direction.z * tmp_dt;
			temp_sub.x = intersect.x - r.origin.x;
			temp_sub.y = intersect.y - r.origin.y;
			temp_sub.z = intersect.z - r.origin.z;
			float3 v2;
			v2.x = intersect.x - t.p1.x; 
			v2.y = intersect.y - t.p1.y;
			v2.z = intersect.z - t.p1.z;
			float dot02 = dot(t.sb21, v2);
			float dot12 = dot(t.sb31, v2);
			float invD = __fdividef(1.0f, (t.dot2121 * t.dot3131 - t.dot2131 * t.dot2131));
			float u = (t.dot3131 * dot02 - t.dot2131 * dot12) * invD;
			float v = (t.dot2121 * dot12 - t.dot2131 * dot02) * invD;
			tmp_dt = dot(temp_sub, r.direction);
			if ((u < 0) || (v < 0) || (u + v > 1) || tmp_dt < 0.0f) { continue; }
			float newDist = matgnitude(temp_sub);
			if (r.hit_triangle && (newDist >= r.last_dist)) {
				continue;
			}
			r.last_dist = newDist;
			r.last_triangle_index = ti;
			r.last_intersect = intersect;
			r.hit_triangle = true;
		}
	}
	if (!r.hit_triangle) { return; }
	r.num_intersects++;
	triangle t = triangles[r.last_triangle_index];
	material m = triangle_materials[r.last_triangle_index];
	float3 nv = t.nv;
	float tmp_d = dot(r.direction, nv);
	r.direction.x = -1 * r.direction.x - nv.x * 2 * tmp_d;
	r.direction.y = -1 * r.direction.y - nv.y * 2 * tmp_d;
	r.direction.z = -1 * r.direction.z - nv.z * 2 * tmp_d;
	r.direction = rand_offset_float3(r.direction, nv, m.roughness, index * iteration);
	r.origin = r.last_intersect;
	rays[index] = r;
	color c = device_color_buffer[index];
	c.r += m.c.r;
	c.g += m.c.g;
	c.b += m.c.b;
	device_color_buffer[index] = c;
	max_brightness_buffer[index] = max_brightness_buffer[index] < m.brightness ? m.brightness : max_brightness_buffer[index];
}

void call_bounce_kernel(int iteration) {
	bounceKernel << <threads_bounce, blocks_bounce >> > (iteration);
}

__global__ void div_colors_kernel() {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	ray r = rays[id];
	if (!r.hit_triangle) { return; }
	float mult = max_brightness_buffer[id] / r.num_intersects;
	device_color_buffer[id].r *= mult;
	device_color_buffer[id].g *= mult;
	device_color_buffer[id].b *= mult;
}

void div_colors() {
	div_colors_kernel << <1024, scr_w* scr_h / 1024 >> > ();
}

ray raysCPU[scr_w * scr_h];

void copy_rays_CPU() {
	hipMemcpyFromSymbol(raysCPU, HIP_SYMBOL(rays), sizeof(raysCPU));
}

void copy_colors_CPU() {
	hipMemcpyFromSymbol(color_buffer, HIP_SYMBOL(device_color_buffer), sizeof(color_buffer));
}



FILE* open_file(char* filename) {
	FILE* ret = fopen(filename, "w");
	if (ret == NULL) {
		printf("%s\n", "error opening file %s\n", filename);
		return NULL;
	}
	return ret;
}

void write_pixel_data_to_txt() {
	unsigned char* pixels;
	FILE* f = open_file("colorReturnFile.txt");
	for (int l = 0; l < scr_w*scr_h; l++) {
		fprintf(f, "%f,%f,%f\n", color_buffer[l].r, color_buffer[l].g, color_buffer[l].b);
	}
	fclose(f);
}

void cycleRays(int iteration) {
	init_rays_call();
	for (int t = 0; t < num_bounces; t++) {
		reset_rays_call();
		call_bounce_kernel(iteration);
	}
	div_colors();
}

int main() {
	clock_t start, end;
	hipDeviceReset();
	init_cube_CPU(make_float3(100.0f, 200.0f, 10.0f), 100.0f, 0, 0.0f, 1.0f, 0.0f, 0.0f, 1.0f);
	start = clock();
	for (int f = 0; f < num_frames; f++) {
		cycleRays(f);
	}
	hipDeviceSynchronize();
	end = clock();
	printf("milis for call: %d\n", end - start);
	copy_rays_CPU();
	copy_colors_CPU();
	write_pixel_data_to_txt();
	hipError_t progErr = hipGetLastError();
	printf("program ended with err: %s\n", hipGetErrorString(progErr));
}